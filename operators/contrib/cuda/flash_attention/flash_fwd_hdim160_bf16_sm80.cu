
#include <hip/hip_runtime.h>
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.
#if USE_FLASH_ATTENTION

#include "flash_fwd_launch_template.h"

namespace flash {

template<>
void run_mha_fwd_<cutlass::bfloat16_t, 160>(Flash_fwd_params& params, cudaStream_t stream) {
    run_mha_fwd_hdim160<cutlass::bfloat16_t>(params, stream);
}

}  // namespace flash
#endif
